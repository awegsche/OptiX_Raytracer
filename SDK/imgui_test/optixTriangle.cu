#include "hip/hip_runtime.h"
//
// Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

// #include <__clang_cuda_runtime_wrapper.h>
#include <optix.h>

#include "camera.h"
#include "optixTriangle.h"
#include "optix_device.h"
#include <cuda/helpers.h>
#include <cuda/random.h>

#include <sutil/vec_math.h>

extern "C" {
__constant__ Params params;
}

static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3 &p)
{
    // Uniformly sample disk.
    const float r   = sqrtf(u1);
    const float phi = 2.0f * M_PIf * u2;
    p.x             = r * cosf(phi);
    p.y             = r * sinf(phi);

    // Project up to hemisphere.
    p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.y * p.y));
}

struct Onb
{
    __forceinline__ __device__ Onb(const float3 &normal)
    {
        m_normal = normal;

        if (fabs(m_normal.x) > fabs(m_normal.z)) {
            m_binormal.x = -m_normal.y;
            m_binormal.y = m_normal.x;
            m_binormal.z = 0;
        } else {
            m_binormal.x = 0;
            m_binormal.y = -m_normal.z;
            m_binormal.z = m_normal.y;
        }

        m_binormal = normalize(m_binormal);
        m_tangent  = cross(m_binormal, m_normal);
    }

    __forceinline__ __device__ void inverse_transform(float3 &p) const
    {
        p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
    }

    float3 m_tangent;
    float3 m_binormal;
    float3 m_normal;
};

static __forceinline__ __device__ void setPayload(float3 p)
{
    optixSetPayload_0(__float_as_uint(p.x));
    optixSetPayload_1(__float_as_uint(p.y));
    optixSetPayload_2(__float_as_uint(p.z));
}

extern "C" __global__ void __raygen__rg()
{
    // Lookup our location within the launch grid
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();


    // Map our launch idx to a screen location and create a ray from the camera
    // location through the screen
    float3 ray_origin, ray_direction;
    // computeRay(params.camera, idx, dim, ray_origin, ray_direction);

    // Trace the ray against our scene hierarchy
    unsigned int       p0, p1, p2;
    const unsigned int index  = idx.y * params.image_width + idx.x;
    float3             result = { 0.0, 0.0, 0.0 };

    unsigned int seed = tea<4>(idx.x + dim.x * idx.y, params.dt);

    for (unsigned int i = 0; i < params.samples_per_frame; ++i) {
        params.camera->compute_ray(idx, dim, ray_origin, ray_direction, seed);
        optixTrace(params.handle,
            ray_origin,
            ray_direction,
            0.0f,// Min intersection distance
            1e16f,// Max intersection distance
            0.0f,// rayTime -- used for motion blur
            OptixVisibilityMask(255),// Specify always visible
            OPTIX_RAY_FLAG_NONE,
            0,// SBT offset   -- See SBT discussion
            1,// SBT stride   -- See SBT discussion
            0,// missSBTIndex -- See SBT discussion
            p0,
            p1,
            p2);
        result.x += __uint_as_float(p0);
        result.y += __uint_as_float(p1);
        result.z += __uint_as_float(p2);
    }

    // Record results in our output raster
    if (params.dirty) {
        params.film[index] = result;
    } else {
        params.film[index] = params.film[index] + result;
    }
    params.image[index] = make_color(params.film[index] / static_cast<float>(params.dt));
}


extern "C" __global__ void __miss__ms()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    // MissData* miss_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    float3 result;
    result.x = static_cast<float>(idx.x) / static_cast<float>(dim.x);
    result.y = static_cast<float>(idx.y) / static_cast<float>(dim.y);
    result.z = static_cast<float>(idx.z) / static_cast<float>(dim.z);
    /*
    unsigned int seed = tea<4>(idx.x + idx.y*dim.x, params.dt);
    const float rand = rnd(seed);
    const float3 result = make_float3(rand, rand, rand);
    */
    setPayload(result);
}


extern "C" __global__ void __closesthit__ch()
{
    // When built-in triangle intersection is used, a number of fundamental
    // attributes are provided by the OptiX API, indlucing barycentric coordinates.

    // TODO: lookup diffuse shading in PBRT / RTFTGU and implement it here.

    // calc normal
    unsigned int vertidx    = optixGetPrimitiveIndex();
    unsigned int vertoffset = vertidx * 3;

    float3 v0 = params.vertices[vertoffset + 1] - params.vertices[vertoffset];
    float3 v1 = params.vertices[vertoffset + 2] - params.vertices[vertoffset];

    float3 normal = normalize(cross(v0, v1));

    const float3 P = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection() + normal * 0.0001f;

    const uint3  idx  = optixGetLaunchIndex();
    const uint3  dim  = optixGetLaunchDimensions();
    unsigned int seed = tea<4>(idx.x + dim.x * idx.y, params.dt);

    float3       result        = { 0.0f, 0.0f, 0.0f };
    const float3 shadow_color  = { 0.0f, 0.0f, 0.0f };
    const float3 ambient_color = { 0.05f, 0.05f, 0.055f };


    // get material
    DiffuseMaterial const &mat = params.materials[params.mat_indices[vertidx]];

    for (int li = 0; li < params.nlights; ++li) {
        const float3 light_wi = params.lights[li].wi(P, seed);
        const float  ndotwi   = dot(normal, light_wi);
        optixTraverse(params.handle,
            P,
            light_wi,
            0.01f,
            1.0f,
            0.0f,
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_CULL_DISABLED_ANYHIT,
            0,
            1,
            0);


        // result += light_color * abs(ndotwi);
        result += (optixHitObjectIsHit() || ndotwi < 0.0f)
                      ? shadow_color
                      : mat.f(P, light_wi, light_wi, seed) * params.lights[li].lumi() * ndotwi;
    }

    Onb onb(normal);

    const float u1 = rnd(seed);
    const float u2 = rnd(seed);
    float3      out;
    cosine_sample_hemisphere(u1, u2, out);

    onb.inverse_transform(out);

    // We are only casting probe rays so no shader invocation is needed
    optixTraverse(params.handle,
        P,
        out,
        0.01,
        1e16f,
        0.0f,// rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0,// SBT offset
        1,// SBT stride
        0// missSBTIndex
    );

    result += optixHitObjectIsHit() ? shadow_color : ambient_color * mat.f(P, out, out, seed);
    setPayload(result);
}
